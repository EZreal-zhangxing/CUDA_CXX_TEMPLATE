#include "hip/hip_runtime.h"
#include "cuda_process.h"

__global__ void print_hello(){
    printf("hello world from gpu block Idx [%d,%d,%d],thread Idx [%d,%d,%d]\n",blockIdx.x,
        blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,blockIdx.z);    
}

/**
 * implement method in cuda_process.h
*/
void print_on_gpu(){
    dim3 block(2,2);
    dim3 grid(5,5);
    print_hello<<<grid,block>>>();
    hipDeviceReset();
}